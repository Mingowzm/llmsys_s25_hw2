#include <hip/hip_runtime.h>
#include <assert.h>
#include <iostream>
#include <sstream>
#include <fstream>

#define MAX_DIMS 10
#define TILE 32
#define BASE_THREAD_NUM 32

#define ADD_FUNC       1
#define MUL_FUNC       2
#define ID_FUNC        3
#define NEG_FUNC       4
#define LT_FUNC        5
#define EQ_FUNC        6
#define SIGMOID_FUNC   7
#define RELU_FUNC      8
#define RELU_BACK_FUNC 9
#define LOG_FUNC       10
#define LOG_BACK_FUNC  11
#define EXP_FUNC       12
#define INV_FUNC       13
#define INV_BACK_FUNC  14
#define IS_CLOSE_FUNC  15
#define MAX_FUNC       16
#define POW            17
#define TANH           18

__device__ float fn(int fn_id, float x, float y=0) {
    switch(fn_id) {
      case ADD_FUNC: {
        return x + y;
      }
      case MUL_FUNC: {
        return x * y;
      }
      case ID_FUNC: {
      	return x;
      }
      case NEG_FUNC: {
        return -x;
      }
      case LT_FUNC: {
        if (x < y) {
          return 1.0;
        }
        else {
          return 0.0;
        }
      }
      case EQ_FUNC: {
        if (x == y) {
          return 1.0;
        }
        else {
          return 0.0;
        }
      }
      case SIGMOID_FUNC: {
        if (x >= 0) {
          return 1.0 / (1.0 + exp(-x));
        }
        else {
          return exp(x) / (1.0 + exp(x));
        }
      }
      case RELU_FUNC: {
        return max(x, 0.0);
      }
      case RELU_BACK_FUNC: {
        if (x > 0) {
          return y;
        }
        else {
          return 0.0;
        }
      }
      case LOG_FUNC: {
        return log(x + 1e-6);
      }
      case LOG_BACK_FUNC: {
        return y / (x + 1e-6);
      }
      case EXP_FUNC: {
        return exp(x);
      }
      case INV_FUNC: {
        return float(1.0 / x);
      }
      case INV_BACK_FUNC: {
        return -(1.0 / (x * x)) * y;
      }
      case IS_CLOSE_FUNC: {
        return (x - y < 1e-2) && (y - x < 1e-2);
      }
      case MAX_FUNC: {
        if (x > y) {
          return x;
        }
        else {
          return y;
        }
      }
      case POW: {
        return pow(x, y);
      }
      case TANH: {
        return tanh(x);
      }
      default: {
        return x + y;
      }
    }

}


__device__ int index_to_position(const int* index, const int* strides, int num_dims) {
    int position = 0;
    for (int i = 0; i < num_dims; ++i) {
        position += index[i] * strides[i];
    }
    return position;
}

__device__ void to_index(int ordinal, const int* shape, int* out_index, int num_dims) {
    int cur_ord = ordinal;
    for (int i = num_dims - 1; i >= 0; --i) {
        int sh = shape[i];
        out_index[i] = cur_ord % sh;
        cur_ord /= sh;
    }
}

__device__ void broadcast_index(const int* big_index, const int* big_shape, const int* shape, int* out_index, int num_dims_big, int num_dims) {
    for (int i = 0; i < num_dims; ++i) {
        if (shape[i] > 1) {
            out_index[i] = big_index[i + (num_dims_big - num_dims)];
        } else {
            out_index[i] = 0;
        }
    }
}


__global__ void MatrixMultiplyKernel(
    float* out,
    const int* out_shape,
    const int* out_strides,
    float* a_storage,
    const int* a_shape,
    const int* a_strides,
    float* b_storage,
    const int* b_shape,
    const int* b_strides
) {
  /**
   * Multiply two (compact) matrices into an output (also comapct) matrix. Matrix a and b are both in a batch
   * format, with shape [batch_size, m, n], [batch_size, n, p].
   * Requirements:
   * - All data must be first moved to shared memory.
   * - Only read each cell in a and b once.
   * - Only write to global memory once per kernel.
   * There is guarantee that a_shape[0] == b_shape[0], a_shape[2] == b_shape[1],
   * and out_shape[0] == a_shape[0], out_shape[1] == b_shape[1]
   *
   * Args:
   *   out: compact 1D array of size batch_size x m x p to write the output to
   *   out_shape: shape of the output array
   *   out_strides: strides of the output array
   *   a_storage: compact 1D array of size batch_size x m x n
   *   a_shape: shape of the a array
   *   a_strides: strides of the a array
   *   b_storage: comapct 2D array of size batch_size x n x p
   *   b_shape: shape of the b array
   *   b_strides: strides of the b array
   *
   * Returns:
   *   None (Fills in out array)
   */

    __shared__ float a_shared[TILE][TILE];
    __shared__ float b_shared[TILE][TILE];

    // In each block, we will compute a batch of the output matrix
    // All the threads in the block will work together to compute this batch
    int batch = blockIdx.z;
    int a_batch_stride = a_shape[0] > 1 ? a_strides[0] : 0;
    int b_batch_stride = b_shape[0] > 1 ? b_strides[0] : 0;
    int m = a_shape[1];
    int n = b_shape[1];
    int p = b_shape[2];

    float accumulator = 0.0f;

    /// BEGIN ASSIGN1_2
    /// TODO
    // Hints:
    // 1. Compute the row and column of the output matrix this block will compute
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    // 2. Compute the position in the output array that this thread will write to
    int out_position = batch * out_strides[0] + row * out_strides[1] + col * out_strides[2];
    // 3. Iterate over tiles of the two input matrices, read the data into shared memory
    int num_tiles = (n + TILE - 1) / TILE;
    for (int i = 0; i < num_tiles; i++) {
      int a_col = i * TILE + threadIdx.y;
      if (row >= m || a_col >= n) {
        a_shared[threadIdx.x][threadIdx.y] = 0.0f;
      } else {
        int a_position = batch * a_batch_stride + row * a_strides[1] + a_col * a_strides[2];
          a_shared[threadIdx.x][threadIdx.y] = a_storage[a_position];
      }
      int b_row = i * TILE + threadIdx.x;
      if (col >= p || b_row >= n) {
        b_shared[threadIdx.x][threadIdx.y] = 0.0f;
      } else {
        int b_pos = batch * b_batch_stride + b_row * b_strides[1] + col * b_strides[2];
        b_shared[threadIdx.x][threadIdx.y] = b_storage[b_pos];
      }
      // 4. Synchronize to make sure the data is available to all threads
      __syncthreads();
      // 5. Compute the output tile for this thread block
      for (int j = 0; j < TILE; j++) {
        accumulator += a_shared[threadIdx.x][j] * b_shared[j][threadIdx.y];
      }
      // 6. Synchronize to make sure all threads are done computing the output tile for (row, col)
      __syncthreads();
    }
    // 7. Write the output to global memory
    if (row < m && col < p) {
        out[out_position] = accumulator;
    }
    // assert(false && "Not Implemented");
    /// END ASSIGN1_2
}


__global__ void mapKernel(
    float* out,
    int* out_shape,
    int* out_strides,
    int out_size,
    float* in_storage,
    int* in_shape,
    int* in_strides,
    int shape_size,
    int fn_id
) {
  /**
   * Map function. Apply a unary function to each element of the input array and store the result in the output array.
   * Optimization: Parallelize over the elements of the output array.
   *
   * You may find the following functions useful:
   * - index_to_position: converts an index to a position in a compact array
   * - to_index: converts a position to an index in a multidimensional array
   * - broadcast_index: converts an index in a smaller array to an index in a larger array
   *
   * Args:
   *  out: compact 1D array of size out_size to write the output to
   *  out_shape: shape of the output array
   *  out_strides: strides of the output array
   *  out_size: size of the output array
   *  in_storage: compact 1D array of size in_size
   *  in_shape: shape of the input array
   *  in_strides: strides of the input array
   *  shape_size: number of dimensions in the input and output arrays, assume dimensions are the same
   *  fn_id: id of the function to apply to each element of the input array
   *
   * Returns:
   *  None (Fills in out array)
   */

    int out_index[MAX_DIMS];
    int in_index[MAX_DIMS];

    /// BEGIN ASSIGN1_2
    /// TODO
    // Hints:
    // 1. Compute the position in the output array that this thread will write to
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= out_size) {
      return;
    }
    // 2. Convert the position to the out_index according to out_shape
    to_index(idx, out_shape, out_index, shape_size);
    // 3. Broadcast the out_index to the in_index according to in_shape (optional in some cases)
    broadcast_index(out_index, out_shape, in_shape, in_index, shape_size, shape_size);
    // 4. Calculate the position of element in in_array according to in_index and in_strides
    int in_position = index_to_position(in_index, in_strides, shape_size);
    // 5. Calculate the position of element in out_array according to out_index and out_strides
    int out_position = index_to_position(out_index, out_strides, shape_size);
    // 6. Apply the unary function to the input element and write the output to the out memory
    out[out_position] = fn(fn_id, in_storage[in_position]);
    // assert(false && "Not Implemented");
    /// END ASSIGN1_2
}


__global__ void reduceKernel(
    float* out,
    int* out_shape,
    int* out_strides,
    int out_size,
    float* a_storage,
    int* a_shape,
    int* a_strides,
    int reduce_dim,
    float reduce_value,
    int shape_size,
    int fn_id
) {
  /**
   * Reduce function. Apply a reduce function to elements of the input array a and store the result in the output array.
   * Optimization:
   * Parallelize over the reduction operation. Each kernel performs one reduction.
   * e.g. a = [[1, 2, 3], [4, 5, 6]], kernel0 computes reduce([1, 2, 3]), kernel1 computes reduce([4, 5, 6]).
   *
   * You may find the following functions useful:
   * - index_to_position: converts an index to a position in a compact array
   * - to_index: converts a position to an index in a multidimensional array
   *
   * Args:
   *  out: compact 1D array of size out_size to write the output to
   *  out_shape: shape of the output array
   *  out_strides: strides of the output array
   *  out_size: size of the output array
   *  a_storage: compact 1D array of size in_size
   *  a_shape: shape of the input array
   *  a_strides: strides of the input array
   *  reduce_dim: dimension to reduce on
   *  reduce_value: initial value for the reduction
   *  shape_size: number of dimensions in the input & output array, assert dimensions are the same
   *  fn_id: id of the reduce function, currently only support add, multiply, and max
   *
   *
   * Returns:
   *  None (Fills in out array)
   */

    // __shared__ double cache[BLOCK_DIM]; // Uncomment this line if you want to use shared memory to store partial results
    int out_index[MAX_DIMS];

    /// BEGIN ASSIGN1_2
    /// TODO
    // 1. Define the position of the output element that this thread or this block will write to
    int thread_id = threadIdx.x + blockIdx.x * blockDim.x;
    if (thread_id >= out_size) {
      return;
    }
    // 2. Convert the out_pos to the out_index according to out_shape
    to_index(thread_id, out_shape, out_index, shape_size);
    int out_position = index_to_position(out_index, out_strides, shape_size);
    // 3. Initialize the reduce_value to the output element
    float reduced = reduce_value;
    // 4. Iterate over the reduce_dim dimension of the input array to compute the reduced value
    for (int i = 0; i < a_shape[reduce_dim]; i++) {
      int input_index[MAX_DIMS];
      for (int j = 0; j < shape_size; j++) {
          input_index[j] = out_index[j];
      }
      input_index[reduce_dim] = i;
      int input_position = index_to_position(input_index, a_strides, shape_size);
      reduced = fn(fn_id, reduced, a_storage[input_position]);
    }
    // 5. Write the reduced value to out memory
    out[out_position] = reduced;
    // assert(false && "Not Implemented");
    /// END ASSIGN1_2

}

__global__ void zipKernel(
    float* out,
    int* out_shape,
    int* out_strides,
    int out_size,
    int out_shape_size,
    float* a_storage,
    int* a_shape,
    int* a_strides,
    int a_shape_size,
    float* b_storage,
    int* b_shape,
    int* b_strides,
    int b_shape_size,
    int fn_id
) {
  /**
   * Zip function. Apply a binary function to elements of the input array a & b and store the result in the output array.
   * Optimization: Parallelize over the elements of the output array.
   *
   * You may find the following functions useful:
   * - index_to_position: converts an index to a position in a compact array
   * - to_index: converts a position to an index in a multidimensional array
   * - broadcast_index: converts an index in a smaller array to an index in a larger array
   *
   * Args:
   *  out: compact 1D array of size out_size to write the output to
   *  out_shape: shape of the output array
   *  out_strides: strides of the output array
   *  out_size: size of the output array
   *  out_shape_size: number of dimensions in the output array
   *  a_storage: compact 1D array of size in_size
   *  a_shape: shape of the input array
   *  a_strides: strides of the input array
   *  a_shape_size: number of dimensions in the input array
   *  b_storage: compact 1D array of size in_size
   *  b_shape: shape of the input array
   *  b_strides: strides of the input array
   *  b_shape_size: number of dimensions in the input array
   *  fn_id: id of the function to apply to each element of the a & b array
   *
   *
   * Returns:
   *  None (Fills in out array)
   */

    int out_index[MAX_DIMS];
    int a_index[MAX_DIMS];
    int b_index[MAX_DIMS];

    /// BEGIN ASSIGN1_2
    /// TODO
    // Hints:
    // 1. Compute the position in the output array that this thread will write to
    int thread_id = threadIdx.x + blockIdx.x * blockDim.x;
    if (thread_id >= out_size) {
      return;
    }

    // 2. Convert the position to the out_index according to out_shape
    to_index(thread_id, out_shape, out_index, out_shape_size);

    // 3. Calculate the position of element in out_array according to out_index and out_strides
    int out_position = index_to_position(out_index, out_strides, out_shape_size);

    // 4. Broadcast the out_index to the a_index according to a_shape
    broadcast_index(out_index, out_shape, a_shape, a_index, out_shape_size, a_shape_size);

    // 5. Calculate the position of element in a_array according to a_index and a_strides
    int a_position = index_to_position(a_index, a_strides, a_shape_size);

    // 6. Broadcast the out_index to the b_index according to b_shape
    broadcast_index(out_index, out_shape, b_shape, b_index, out_shape_size, b_shape_size);

    // 7.Calculate the position of element in b_array according to b_index and b_strides
    int b_position = index_to_position(b_index, b_strides, b_shape_size);

    // 8. Apply the binary function to the input elements in a_array & b_array and write the output to the out memory
    out[out_position] = fn(fn_id, a_storage[a_position], b_storage[b_position]);

    // assert(false && "Not Implemented");
    /// END ASSIGN1_2
}


extern "C" {

void MatrixMultiply(
    float* out,
    int* out_shape,
    int* out_strides,
    float* a_storage,
    int* a_shape,
    int* a_strides,
    float* b_storage,
    int* b_shape,
    int* b_strides,
    int batch, int m, int p
) {
    int n = a_shape[2];

    // Allocate device memory
    float *d_out, *d_a, *d_b;
    hipMalloc(&d_a, batch * m * n * sizeof(float));
    hipMalloc(&d_b, batch * n * p * sizeof(float));
    hipMalloc(&d_out, batch * m * p * sizeof(float));

    int *d_out_shape, *d_out_strides, *d_a_shape, *d_a_strides, *d_b_shape, *d_b_strides;
    hipMalloc(&d_out_shape, 3 * sizeof(int));
    hipMalloc(&d_out_strides, 3 * sizeof(int));
    hipMalloc(&d_a_shape, 3 * sizeof(int));
    hipMalloc(&d_a_strides, 3 * sizeof(int));
    hipMalloc(&d_b_shape, 3 * sizeof(int));
    hipMalloc(&d_b_strides, 3 * sizeof(int));


    // Copy data to the device
    hipMemcpy(d_a, a_storage, batch * m * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b_storage, batch * n * p * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_out_shape, out_shape, 3 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_out_strides, out_strides, 3 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_a_shape, a_shape, 3 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_a_strides, a_strides, 3 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b_shape, b_shape, 3 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b_strides, b_strides, 3 * sizeof(int), hipMemcpyHostToDevice);

    int threadsPerBlock = BASE_THREAD_NUM;
    dim3 blockDims(threadsPerBlock, threadsPerBlock, 1); // Adjust these values based on your specific requirements
    dim3 gridDims((m + threadsPerBlock - 1) / threadsPerBlock, (p + threadsPerBlock - 1) / threadsPerBlock, batch);
    MatrixMultiplyKernel<<<gridDims, blockDims>>>(
        d_out, d_out_shape, d_out_strides, d_a, d_a_shape, d_a_strides, d_b, d_b_shape, d_b_strides
    );

    // Copy back to the host
    hipMemcpy(out, d_out, batch * m * p * sizeof(float), hipMemcpyDeviceToHost);

    hipDeviceSynchronize();

    // Check CUDA execution
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
      fprintf(stderr, "Matmul Error: %s\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }

    // Free memory on device
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);
    hipFree(d_out_shape);
    hipFree(d_out_strides);
    hipFree(d_a_shape);
    hipFree(d_a_strides);
    hipFree(d_b_shape);
    hipFree(d_b_strides);
}

void tensorMap(
    float* out,
    int* out_shape,
    int* out_strides,
    int out_size,
    float* in_storage,
    int* in_shape,
    int* in_strides,
    int in_size,
    int shape_size,
    int fn_id
) {

    float *d_out, *d_in;
    hipMalloc(&d_out, out_size * sizeof(float));
    hipMalloc(&d_in, in_size * sizeof(float));

    int *d_out_shape, *d_out_strides, *d_in_shape, *d_in_strides;
    hipMalloc(&d_out_shape, shape_size * sizeof(int));
    hipMalloc(&d_out_strides, shape_size * sizeof(int));
    hipMalloc(&d_in_shape, shape_size * sizeof(int));
    hipMalloc(&d_in_strides, shape_size * sizeof(int));

    hipMemcpy(d_in, in_storage, in_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_out_shape, out_shape, shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_out_strides, out_strides, shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_in_shape, in_shape, shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_in_strides, in_strides, shape_size * sizeof(int), hipMemcpyHostToDevice);

    int threadsPerBlock = BASE_THREAD_NUM;
    int blocksPerGrid = (out_size + threadsPerBlock - 1) / threadsPerBlock;
    mapKernel<<<blocksPerGrid, threadsPerBlock>>>(
      d_out, d_out_shape, d_out_strides, out_size,
      d_in, d_in_shape, d_in_strides,
      shape_size, fn_id);

    // Copy back to the host
    hipMemcpy(out, d_out, out_size * sizeof(float), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    // Check CUDA execution
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
      fprintf(stderr, "Map Error: %s\n", hipGetErrorString(err));
      // Handle the error (e.g., by exiting the program)
      exit(EXIT_FAILURE);
    }

    // Free memory on device
    hipFree(d_in);
    hipFree(d_out);
    hipFree(d_out_shape);
    hipFree(d_out_strides);
    hipFree(d_in_shape);
    hipFree(d_in_strides);
}


void tensorZip(
    float* out,
    int* out_shape,
    int* out_strides,
    int out_size,
    int out_shape_size,
    float* a_storage,
    int* a_shape,
    int* a_strides,
    int a_size,
    int a_shape_size,
    float* b_storage,
    int* b_shape,
    int* b_strides,
    int b_size,
    int b_shape_size,
    int fn_id
) {

    // Allocate device memory
    float *d_out, *d_a, *d_b;
    hipMalloc((void **)&d_a, a_size * sizeof(float));
    hipMalloc(&d_b, b_size * sizeof(float));
    hipMalloc(&d_out, out_size * sizeof(float));

    int *d_out_shape, *d_out_strides, *d_a_shape, *d_a_strides, *d_b_shape, *d_b_strides;
    hipMalloc(&d_out_shape, out_shape_size * sizeof(int));
    hipMalloc(&d_out_strides, out_shape_size * sizeof(int));
    hipMalloc(&d_a_shape, a_shape_size * sizeof(int));
    hipMalloc(&d_a_strides, a_shape_size * sizeof(int));
    hipMalloc(&d_b_shape, b_shape_size * sizeof(int));
    hipMalloc(&d_b_strides, b_shape_size * sizeof(int));

    // Copy data to the device
    hipMemcpy(d_a, a_storage, a_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b_storage, b_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_out_shape, out_shape, out_shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_out_strides, out_strides, out_shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_a_shape, a_shape, a_shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_a_strides, a_strides, a_shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b_shape, b_shape, b_shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b_strides, b_strides, b_shape_size * sizeof(int), hipMemcpyHostToDevice);

    // Launch kernel
    int threadsPerBlock = BASE_THREAD_NUM;
    int blocksPerGrid = (out_size + threadsPerBlock - 1) / threadsPerBlock;
    zipKernel<<<blocksPerGrid, threadsPerBlock>>>(
      d_out, d_out_shape, d_out_strides, out_size, out_shape_size,
      d_a, d_a_shape, d_a_strides, a_shape_size,
      d_b, d_b_shape, d_b_strides, b_shape_size,
      fn_id);

    // Copy back to the host
    hipMemcpy(out, d_out, out_size * sizeof(float), hipMemcpyDeviceToHost);

    hipDeviceSynchronize();


    // Check CUDA execution
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
      fprintf(stderr, "Zip Error: %s\n", hipGetErrorString(err));
      // Handle the error (e.g., by exiting the program)
      exit(EXIT_FAILURE);
    }

    // Free memory on device
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);
    hipFree(d_out_shape);
    hipFree(d_out_strides);
    hipFree(d_a_shape);
    hipFree(d_a_strides);
    hipFree(d_b_shape);
    hipFree(d_b_strides);
}



void tensorReduce(
    float* out,
    int* out_shape,
    int* out_strides,
    int out_size,
    float* a_storage,
    int* a_shape,
    int* a_strides,
    int reduce_dim,
    float reduce_value,
    int shape_size,
    int fn_id
) {
    int a_size = out_size * a_shape[reduce_dim];
    float *d_out, *d_a;
    hipMalloc(&d_out, out_size * sizeof(float));
    hipMalloc(&d_a, a_size * sizeof(float));

    int *d_out_shape, *d_out_strides, *d_a_shape, *d_a_strides;
    hipMalloc(&d_out_shape, shape_size * sizeof(int));
    hipMalloc(&d_out_strides, shape_size * sizeof(int));
    hipMalloc(&d_a_shape, shape_size * sizeof(int));
    hipMalloc(&d_a_strides, shape_size * sizeof(int));

    hipMemcpy(d_a, a_storage, a_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_out_shape, out_shape, shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_out_strides, out_strides, shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_a_shape, a_shape, shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_a_strides, a_strides, shape_size * sizeof(int), hipMemcpyHostToDevice);

    int threadsPerBlock = BASE_THREAD_NUM;
    int blocksPerGrid = (out_size + threadsPerBlock - 1) / threadsPerBlock;
    reduceKernel<<<blocksPerGrid, threadsPerBlock>>>(
        d_out, d_out_shape, d_out_strides, out_size,
        d_a, d_a_shape, d_a_strides,
        reduce_dim, reduce_value, shape_size, fn_id
    );

    // Copy back to the host
    hipMemcpy(out, d_out, out_size * sizeof(float), hipMemcpyDeviceToHost);

    hipDeviceSynchronize();

    // Check CUDA execution
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
      fprintf(stderr, "Reduce Error: %s\n", hipGetErrorString(err));
      // Handle the error (e.g., by exiting the program)
      exit(EXIT_FAILURE);
    }

    hipFree(d_a);
    hipFree(d_out);
    hipFree(d_out_shape);
    hipFree(d_out_strides);
    hipFree(d_a_shape);
    hipFree(d_a_strides);
}

}